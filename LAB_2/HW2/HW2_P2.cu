
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start,0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

__global__ void addVecKernel(int *in1, int *in2, int n, 
        int *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if (i < n)
    {
        out[i] = in1[i] + in2[i];
    }
}

void addVec(int *in1, int *in2, int n, 
        int *out, 
        bool useDevice=false, dim3 blockSize=dim3(1), int nStreams=1)
{
	if (useDevice == false)
	{
        for (int i = 0; i < n; i++)
        {
            out[i] = in1[i] + in2[i];
        }
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);
        
        // Pin host memory regions (allocated by malloc)
        // so that we can use cudaMemcpyAsync  
        size_t nBytes = n * sizeof(int);
        CHECK(hipHostRegister(in1, nBytes, hipHostRegisterDefault));
        CHECK(hipHostRegister(in2, nBytes, hipHostRegisterDefault));
        CHECK(hipHostRegister(out, nBytes, hipHostRegisterDefault));

		// TODO: Allocate device memory regions

        // TODO: Create "nStreams" device streams

        GpuTimer timer;
        timer.Start();

        // TODO: Send jobs (H2D, kernel, D2H) to device streams 

        timer.Stop();
        float time = timer.Elapsed();
        printf("Processing time of all device streams: %f ms\n\n", time);

        // TODO: Destroy device streams

        // TODO: Free device memory regions

        // Unpin host memory regions
        CHECK(hipHostUnregister(in1));
        CHECK(hipHostUnregister(in2));
        CHECK(hipHostUnregister(out));
	}
}

int main(int argc, char ** argv)
{
    int n; 
    int *in1, *in2; 
    int *out, *correctOut;

    // Input data into n
    n = (1 << 24) + 1;
    printf("n =  %d\n\n", n);

    // Allocate memories for in1, in2, out
    size_t nBytes = n * sizeof(int);
    in1 = (int *)malloc(nBytes);
    in2 = (int *)malloc(nBytes);
    out = (int *)malloc(nBytes);
    /*
    CHECK(cudaMallocHost(&in1, nBytes));
    CHECK(cudaMallocHost(&in2, nBytes));
    CHECK(cudaMallocHost(&out, nBytes));
    */
    correctOut = (int *)malloc(nBytes);

    // Input data into in1, in2
    for (int i = 0; i < n; i++)
    {
    	in1[i] = rand() & 0xff; // Random int in [0, 255]
    	in2[i] = rand() & 0xff; // Random int in [0, 255]
    }

    // Add in1 & in2 on host
    addVec(in1, in2, n, correctOut);

    // Add in1 & in2 on device
	dim3 blockSize(512); // Default
    int nStreams = 1; // Default
	if (argc >= 2)
	{
		blockSize.x = atoi(argv[1]);
        if (argc >= 3)
        {
            nStreams = atoi(argv[2]);
        }
	} 
    addVec(in1, in2, n, out, true, blockSize, nStreams);

    // Check correctness
    for (int i = 0; i < n; i++)
    {
    	if (out[i] != correctOut[i])
    	{
    		printf("INCORRECT :(\n");
    		return 1;
    	}
    }
    printf("CORRECT :)\n");

    free(in1);
    free(in2);
    free(out);
    /*
    CHECK(cudaFreeHost(in1));
    CHECK(cudaFreeHost(in2));
    CHECK(cudaFreeHost(out));
    */
    free(correctOut);
}
