// Last update: 16/12/2020

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(1);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// Sequential Radix Sort
void sortByHost(const uint32_t * in, int n,
                uint32_t * out)
{
    int * bits = (int *)malloc(n * sizeof(int));
    int * nOnesBefore = (int *)malloc(n * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // To free memory later
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    // Loop from LSB (Least Significant Bit) to MSB (Most Significant Bit)
	// In each loop, sort elements according to the current bit from src to dst 
	// (using STABLE counting sort)
    for (int bitIdx = 0; bitIdx < sizeof(uint32_t) * 8; bitIdx++)
    {
        // Extract bits
        for (int i = 0; i < n; i++)
            bits[i] = (src[i] >> bitIdx) & 1;

        // Compute nOnesBefore
        nOnesBefore[0] = 0;
        for (int i = 1; i < n; i++)
            nOnesBefore[i] = nOnesBefore[i-1] + bits[i-1];

        // Compute rank and write to dst
        int nZeros = n - nOnesBefore[n-1] - bits[n-1];
        for (int i = 0; i < n; i++)
        {
            int rank;
            if (bits[i] == 0)
                rank = i - nOnesBefore[i];
            else
                rank = nZeros + nOnesBefore[i];
            dst[rank] = src[i];
        }

        // Swap src and dst
        uint32_t * temp = src;
        src = dst;
        dst = temp;
    }

    // Does out array contain results?
    memcpy(out, src, n * sizeof(uint32_t));

    // Free memory
    free(originalSrc);
    free(bits);
    free(nOnesBefore);
}

// Parallel Radix Sort
void sortByDevice(const uint32_t * in, int n, uint32_t * out, int blockSize)
{
    // TODO

}

// Radix Sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        bool useDevice=false, int blockSize=1)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix Sort by host\n");
        sortByHost(in, n, out);
    }
    else // use device
    {
    	printf("\nRadix Sort by device\n");
        sortByDevice(in, n, out, blockSize);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    //int n = 50; // For test by eye
    int n = (1 << 24) + 1;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
    {
        //in[i] = rand() % 255; // For test by eye
        in[i] = rand();
    }
    //printArray(in, n); // For test by eye

    // DETERMINE BLOCK SIZE
    int blockSize = 512; // Default 
    if (argc == 2)
        blockSize = atoi(argv[1]);

    // SORT BY HOST
    sort(in, n, correctOut);
    //printArray(correctOut, n); // For test by eye
    
    // SORT BY DEVICE
    sort(in, n, out, true, blockSize);
    //printArray(out, n); // For test by eye
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
