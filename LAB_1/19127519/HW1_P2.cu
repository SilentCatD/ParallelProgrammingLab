
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>


#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel(uchar3 * inPixels, int width, int height, 
		float * filter, int filterWidth, 
		uchar3 * outPixels)
{
	// TODO

	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if (c < width && r < height) {
		int i = r * width + c;
		float3 sum;
		sum.x = 0;
		sum.y = 0;
		sum.z = 0;
		for (int r_filter = 0; r_filter < filterWidth; r_filter++) {
			for (int c_filter = 0; c_filter < filterWidth; c_filter++) {
				int currentPixelR = r - (filterWidth / 2) + r_filter;
				int currentPixelC = c - (filterWidth / 2) + c_filter;
				if (currentPixelR < 0) {
					currentPixelR = 0;
				}
				if (currentPixelR > height) {
					currentPixelR = height - 1;
				}
				if (currentPixelC < 0) {
					currentPixelC = 0;
				}
				if (currentPixelR > width) {
					currentPixelR = width - 1;
				}
				int pixelIdx = currentPixelR * width + currentPixelC;
				int filterIdx = r_filter * filterWidth + c_filter;
				sum.x += inPixels[pixelIdx].x * filter[filterIdx];
				sum.y += inPixels[pixelIdx].y * filter[filterIdx];
				sum.z += inPixels[pixelIdx].z * filter[filterIdx];
			}
		}
		uchar3 result;
		result.x = sum.x;
		result.y = sum.y;
		result.z = sum.z;
		outPixels[i] = result;
		}

}
	

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO
		for (int r = 0; r < width; r++) {
			for (int c = 0; c < height; c++) {
				int i = r * width + c;
				float3 sum;
				sum.x = 0;
				sum.y = 0;
				sum.z = 0;
				for (int r_filter = 0; r_filter < filterWidth; r_filter++) {
					for (int c_filter = 0; c_filter < filterWidth; c_filter++) {
						int currentPixelR = r - (filterWidth / 2) + r_filter;
						int currentPixelC = c - (filterWidth / 2) + c_filter;
						if (currentPixelR < 0) {
							currentPixelR = 0;
						}
						if (currentPixelR > height) {
							currentPixelR = height - 1;
						}
						if (currentPixelC < 0) {
							currentPixelC = 0;
						}
						if (currentPixelR > width) {
							currentPixelR = width - 1;
						}
						int pixelIdx = currentPixelR * width + currentPixelC;
						int filterIdx = r_filter * filterWidth + c_filter;
						sum.x += inPixels[pixelIdx].x * filter[filterIdx];
						sum.y += inPixels[pixelIdx].y * filter[filterIdx];
						sum.z += inPixels[pixelIdx].z * filter[filterIdx];
					}
				}
				uchar3 result;
				result.x = sum.x;
				result.y = sum.y;
				result.z = sum.z;
				outPixels[i] = result;
			}
		}

	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO
		// TODO: Allocate device memories
		uchar3 *d_inPixels, *d_outPixels;
		float* d_filter;
		size_t nBytesUchar = width * height * sizeof(uchar3);
		size_t nBytesFloat = filterWidth* filterWidth * sizeof(float);
		CHECK(hipMalloc(&d_inPixels, nBytesUchar));
		CHECK(hipMalloc(&d_outPixels, nBytesUchar));
		CHECK(hipMalloc(&d_filter, nBytesFloat));
		// TODO: Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, nBytesUchar, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_filter, filter, nBytesFloat, hipMemcpyHostToDevice));
		// TODO: Set grid size and call kernel (remember to check kernel error)
		int maxThread = devProp.maxThreadsPerBlock;
		int totalThread = blockSize.x * blockSize.y;
		if (totalThread > maxThread) {
			printf("\n=====================================\n");
			printf("ERROR: Can't allocate more than %i threads per block; got: %i\n", maxThread, totalThread);
			exit(EXIT_FAILURE);
		}
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		blurImgKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter,filterWidth,d_outPixels);
		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, nBytesUchar, hipMemcpyDeviceToHost));
		// TODO: Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));
		CHECK(hipFree(d_filter));

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect 
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);
	
	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}  
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
